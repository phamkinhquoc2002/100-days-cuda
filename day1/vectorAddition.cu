#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(float *A, float *B, float *C, int N) {
    int idx = blockIdx.x * blockDim.x+ threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    float milliseconds = 0;

    int N = 1024;
    const int size = N * sizeof(int);
    float *d_A; 
    float *d_B;
    float *d_C;
    

    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    for (int i=0; i<N; i++) {
        h_A[i] = 1;
        h_B[i] = 1;
    };

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_A, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = N / threadsPerBlock;

    vectorAdd<<<blocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU Time: %.4f ms\n", milliseconds);


    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    for (int i =0; i < 5; i ++) {
        std::cout << "C[" << i <<"]"<<"is" << h_C[i] << std::endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;

}