#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matMul(float* A, float* B, float* C, int M, int N, int K) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if(row < M && col < N) {
        float val = 0.0f;
        for (int k = 0; k < K; k++) {
            val += A[row*K + k] * B[k*N + col];
        }
        C[row * N + col] = val;
    }
}

int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    float milliseconds = 0;

    int M = 128;
    int N = 256;
    int K = 512;

    float *A, *B, *C;

    A = (float *)malloc(M*K*sizeof(float));
    B = (float *)malloc(K*N*sizeof(float));
    C = (float *)malloc(M*N*sizeof(float));

    for (int i=0; i < M; i ++) {
        for (int j=0; j < K; j ++) {
            A[i * K + j] = 1.0f;
        }
    }

    for (int i=0; i < K; i ++) {
        for (int j=0; j < N; j ++) {
            B[i * N + j] = 1.0f;
        }
    }

    float *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, M * K * sizeof(float));
    hipMalloc((void**)&d_B, K * N * sizeof(float));
    hipMalloc((void**)&d_C, M * N * sizeof(float));

    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim(ceil(N/16.0f), ceil(M/16.0f));

    matMul<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, N, K);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    printf("GPU Time: %.4f ms\n", milliseconds);
    hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_C);
    hipFree(d_B);
}


